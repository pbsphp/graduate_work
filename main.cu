#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#include "des.h"


int main()
{
    const int TOTAL_DATA = 1005000;

    uint64_t *data = (uint64_t *) malloc(TOTAL_DATA * sizeof(uint64_t));

    for (int i = 0; i < TOTAL_DATA; ++i) {
        data[i] = 0xDEFECA7ED1C0FFEE;
    }

    // const uint64_t key = 0xDEADFACEDEADFACE;

    // for (int i = 0; i < 500; ++i) {
    //     des_encrypt(data, TOTAL_DATA, key);
    //     des_decrypt(data, TOTAL_DATA, key);
    // }

    const uint64_t keys[3] = { 0x12345678ABCDEF00, 0xDEADFACEDEADFACE, 0xDEADBEEFDEADBEEF };

    for (int i = 0; i < 500; ++i) {
        tdes_ede_encrypt(data, TOTAL_DATA, keys);
        tdes_ede_decrypt(data, TOTAL_DATA, keys);
    }

    for (int i = 0; i < 5; ++i) {
        printf("%lx\n", data[i]);
    }

    printf("%d\n", hipGetLastError());

    free(data);

    return 0;
}
